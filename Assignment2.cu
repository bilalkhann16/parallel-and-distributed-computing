#include<stdio.h>
#include<hip/hip_runtime.h>
#include <time.h>

//Subhan Khaliq
//P180095

__global__ void odd_even_sort(int* x,int I,int n)
{
        int id=blockIdx.x;
        if(I==0 && ((id*2+1)< n)){
                if(x[id*2]>x[id*2+1]){
                        int X=x[id*2];
                        x[id*2]=x[id*2+1];
                        x[id*2+1]=X;
                }
        }
        if(I==1 && ((id*2+2)< n)){
                if(x[id*2+1]>x[id*2+2]){
                        int X=x[id*2+1];
                        x[id*2+1]=x[id*2+2];
                        x[id*2+2]=X;
                }
        }
}
void populate(int array[], int n){
    printf("Populate the Array with Random number between 0-10");
    for(int i = 0; i < n; i++){
        array[i] = (rand() %10) +1;
    }
}
int main()
{
        int array[100],n,c[100],i;
        int *d;
	double time_spent = 0.0;

        printf("Enter size of the Array : ");
        scanf("%d",&n);
        //Populate Array with Random Numbers
        populate(array,n);
        hipMalloc((void**)&d, n*sizeof(int));

        hipMemcpy(d,array,n*sizeof(int),hipMemcpyHostToDevice);
        //Time Spend While Execution
        clock_t begin = clock();
        for(i=0;i<n;i++){

                //int size=n/2;

                odd_even_sort<<<n/2,1>>>(d,i%2,n);
        }
	clock_t end = clock();
        printf("\n");


        hipMemcpy(c,d,n*sizeof(int), hipMemcpyDeviceToHost);
        printf("Sorted Array is:\t");
        for(i=0; i<n; i++)
        {
                printf("%d\t",c[i]);
        }
        printf("\n");
        time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
        printf("The time is %f seconds", time_spent);
        printf("\n");
        hipFree(d);
        return 0;
}
